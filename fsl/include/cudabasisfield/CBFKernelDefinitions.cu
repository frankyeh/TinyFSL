#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////////////
/// \file
/// \brief Implementation of cuda-specific code
/// \details Specifically CBFKernelHelpers and CBFKernels functions
/// \author Frederik Lange
/// \date February 2018
/// \copyright Copyright (C) 2018 University of Oxford
//////////////////////////////////////////////////////////////////////////////////////////////

#include "CBFKernelHelpers.cuh"
#include "CBFKernels.cuh"

#include <hip/hip_runtime.h>

#include <vector>

// CBFKernelHelpers Definitions
namespace CBF
{
    /// Convert from volume coordinates to linear coordinates
    __host__ __device__ void index_vol_to_lin(
                                // Input
                                // Vol coords
                                unsigned int xind, unsigned int yind, unsigned int zind,
                                // Vol dims
                                unsigned int szx, unsigned int szy, unsigned int szz,
                                // Output
                                // Index into linear array
                                unsigned int *lind)
    {
        *lind = zind*(szx*szy) + yind*(szx) + xind;
    }

    /// Convert from linear coordinates to volume coordinates
    __host__ __device__ void index_lin_to_vol(
                                // Input
                                // Index into linear array
                                unsigned int lind,
                                // Vol dims
                                unsigned int szx, unsigned int szy, unsigned int szz,
                                 // Output
                                unsigned int *xind, unsigned int *yind, unsigned int *zind)
    {
        *zind = lind/(szx*szy);
        *yind = (lind - *zind*(szx*szy))/szx;
        *xind = lind - *zind*(szx*szy) - *yind*(szx);
    }

    /// Convert from linear coordinates back to linear coordinates of different spaces
    __host__ __device__ void index_lin_to_lin(
                                // Input
                                // Original linear index
                                unsigned int lind_in,
                                // size of original volume space
                                unsigned int lv_szx, unsigned int lv_szy, unsigned int lv_szz,
                                // size of new volume space
                                unsigned int vl_szx, unsigned int vl_szy, unsigned int vl_szz,
                                // Output
                                unsigned int *lind_out)
    {
        // Intermediate variables
        unsigned int xind = 0;
        unsigned int yind = 0;
        unsigned int zind = 0;

        // Convert to original volume space coordinates
        index_lin_to_vol(lind_in,lv_szx,lv_szy,lv_szz,&xind,&yind,&zind);
        index_vol_to_lin(xind,yind,zind,vl_szx,vl_szy,vl_szz,lind_out);
    }

    /// Calculate the start and end row & column for each diagonal in the Hessian
    __host__ __device__ void identify_diagonal(
                                // Input
                                // Index of current diagonal
                                unsigned int diag_ind,
                                // No. of overlapping splines in each direction
                                unsigned int rep_x, unsigned int rep_y, unsigned int rep_z,
                                // Total no. of splines in each direction
                                unsigned int spl_x, unsigned int spl_y, unsigned int spl_z,
                                // Output
                                unsigned int *first_row, unsigned int *last_row,
                                unsigned int *first_column, unsigned int *last_column)
    {
        // Reference variables
        unsigned int hess_side_length = spl_x*spl_y*spl_z;

        // Ensure the main diagonal value is valid
        unsigned int main_diag_lind = (rep_x*rep_y*rep_z-1)/2;
        unsigned int hess_main_diag_lind = 0;
        unsigned int hess_lind = 0;
        // Calculate linear index into hessian
        index_lin_to_lin(main_diag_lind,rep_x,rep_y,rep_z,spl_x,spl_y,spl_z,&hess_main_diag_lind);
        index_lin_to_lin(diag_ind,rep_x,rep_y,rep_z,spl_x,spl_y,spl_z,&hess_lind);
        // Deal with below main diagonal
        if (diag_ind < main_diag_lind)
        {
            *first_row = hess_main_diag_lind - hess_lind;
            *last_row = hess_side_length - 1;
            *first_column = 0;
            *last_column = hess_side_length - *first_row - 1;
        }

        // Deal with main diagonal and above
        else if (diag_ind >= main_diag_lind)
        {
            *first_column = hess_lind - hess_main_diag_lind;
            *last_column = hess_side_length - 1;
            *first_row = 0;
            *last_row = hess_side_length - *first_column - 1;
        }
    }

    /// Get the dimensions of the spline field coeffients based on warp parameterisaton and
    /// image size
    __host__ std::vector<unsigned int> get_spl_coef_dim(const std::vector<unsigned int>& ksp,
                                                        const std::vector<unsigned int>& isz)
    {
      std::vector<unsigned int> rval(ksp.size());
      for (unsigned int i=0; i<ksp.size(); i++)
      {
          rval[i] = static_cast<unsigned int>(std::ceil(float(isz[i]+1) / float(ksp[i]))) + 2;
      }
      return(rval);
    }
} // namespace CBF

//////////////////////////////////////////////////////////////////////////////////////////////
// Functions purely for use in JtJ calculations
//////////////////////////////////////////////////////////////////////////////////////////////
namespace CBF
{
  __device__ void calculate_diagonal_range(
      // Input
      const int offset,
      const unsigned int n_rows,
      const unsigned int n_cols,
      // Output
      unsigned int *first_row,
      unsigned int *last_row,
      unsigned int *first_col,
      unsigned int *last_col)
  {
    // Below main diagonal
    if (offset < 0){
      *first_row = -offset;
      *last_row = n_rows -1;
      *first_col = 0;
      *last_col = n_cols + offset -1;
    }
    // On or above main diagonal
    else{
      *first_row = 0;
      *last_row = n_rows - offset -1;
      *first_col = offset;
      *last_col = n_cols -1;
    }
  }

  __device__ void calculate_overlap(
      // Input
      const int diff_mid,
      const unsigned int ksp,
      const int spl_order,
      // Output
      unsigned int *spl1_start,
      unsigned int *spl1_end,
      unsigned int *spl2_start)
  {
    // spl1 left of spl2
    if (diff_mid < 0){
      *spl1_start = static_cast<unsigned int>(-diff_mid)*ksp;
      *spl1_end = (ksp * (spl_order + 1)) - 2;
      *spl2_start = 0;
    }
    // spl1 right of spl2 or total overlap
    else{
      *spl1_start = 0;
      *spl1_end = (ksp * (spl_order + 1)) - 2 - (diff_mid * ksp);
      *spl2_start = diff_mid * ksp;
    }
  } // calculate_overlap

  __device__ bool is_valid_index(
      const int index,
      const unsigned int n_vals)
  {
    if (index < 0) return false;
    else if (static_cast<unsigned int>(index) >= n_vals) return false;
    else return true;
  } // is_valid_index
} // namespace CBF

//////////////////////////////////////////////////////////////////////////////////////////////
// HessianKernels Definitions
//////////////////////////////////////////////////////////////////////////////////////////////
namespace CBF
{
  ////////////////////////////////////////////////////////////////////////////////////////////
  /// \details This is the most important part of the code so far. We launch a single Kernel
  /// which is responsible for calculating the Hessian. The first iteration of this code was
  /// monolithic and really hard to follow, but I was paranoid about performance. This time I
  /// I am trying to do a much better job of splitting things up into useful functions. Also,
  /// I will try and give the compiler at least a passable shot at optimising by declaring
  /// everything inline!
  ///
  /// An important aspect of this kernel is the logic regarding block and thread ids. They
  /// work as follows:
  ///
  /// gridDim.x  = total number of diagonals being calculated (343 if no symmetry used)
  /// gridDim.y  = total number of chunks each diagonal is broken into
  /// blockIdx.x = number of the current diagonal in jtj, starting at 0 for the lowest
  ///              diagonal present (343 total for cubic spline, therefore main diagonal has
  ///              blockIdx.x == 171)
  /// blockIdx.y = number of the current chunk within the current diagonal
  /// blockDim.x = total number of threads launched for each block (i.e. the size of each chunk
  ///              within each diagonal)
  ///
  /// \param ima_sz_x size of image in x-direction
  /// \param ima_sz_y size of image in y-direction
  /// \param ima_sz_z size of image in z-direction
  /// \param ima pre-multiplied image volume
  /// \param spl_x 1D spline kernel in x-direction
  /// \param spl_y 1D spline kernel in y-direction
  /// \param spl_z 1D spline kernel in z-direction
  /// \param spl_ksp_x knot spacing of spline in x-direction
  /// \param spl_ksp_y knot spacing of spline in y-direction
  /// \param spl_ksp_z knot spacing of spline in z-direction
  /// \param param_sz_x size of parameter space in x-direction
  /// \param param_sz_y size of parameter space in y-direction
  /// \param param_sz_z size of parameter space in z-direction
  /// \param jtj_offsets offsets of sparse diagonal representation of jtj
  ///
  /// \param jtj_values linearise values of sparse diagonal representation of jtj
  ////////////////////////////////////////////////////////////////////////////////////////////
  __global__ void kernel_make_jtj_symmetrical(
      // Input
      unsigned int ima_sz_x,
      unsigned int ima_sz_y,
      unsigned int ima_sz_z,
      hipTextureObject_t ima,
      const float* __restrict__ spl_x,
      const float* __restrict__ spl_y,
      const float* __restrict__ spl_z,
      unsigned int spl_ksp_x,
      unsigned int spl_ksp_y,
      unsigned int spl_ksp_z,
      unsigned int param_sz_x,
      unsigned int param_sz_y,
      unsigned int param_sz_z,
      const int* __restrict__ jtj_offsets,
      // Output
      float* __restrict__ jtj_values)
  {
    __shared__ unsigned int diag_first_row;
    __shared__ unsigned int diag_last_row;
    __shared__ unsigned int diag_first_col;
    __shared__ unsigned int diag_last_col;
    extern __shared__ float all_splines[];
    const int spl_order = 3; // Might replace this with a parameter later
    const auto jtj_sz_diag = param_sz_x * param_sz_y * param_sz_z;
    const auto offset_into_diag = blockIdx.y*blockDim.x + threadIdx.x;
    float *shared_spline_x = all_splines;
    float *shared_spline_y = &all_splines[spl_ksp_x * (spl_order + 1) - 1];
    float *shared_spline_z = &shared_spline_y[spl_ksp_y * (spl_order + 1) - 1];
    // We only need to calculate the valid spline indices once per diagonal
    // This could potentially be done outside of the kernel in fact, but would require a
    // a fairly major reworking of the logic involved, but still potentially worth it.
    __shared__ unsigned int spl1_xind_start, spl1_xind_end, spl2_xind_start;
    __shared__ unsigned int spl1_yind_start, spl1_yind_end, spl2_yind_start;
    __shared__ unsigned int spl1_zind_start, spl1_zind_end, spl2_zind_start;

    // Calculate the overlapping regions of each spline. This only needs to be calculated
    // once per block, as all threads within a block are characterised by the same type of
    // overlap (except when the splines reach the end of the image in any dimension)
    // NB!!! Calculating the overlap once per diagonal only works if the diagonal starts
    // with a valid overlap!!! I.E. this will not work when symmetry is does not hold!!!
    if (threadIdx.x == 0){
      // Identify the first valid (row,col) pair for this particular block
      auto this_offset = jtj_offsets[blockIdx.x];
      CBF::calculate_diagonal_range(
          // Input
          this_offset,
          jtj_sz_diag,
          jtj_sz_diag,
          // Output
          &diag_first_row,
          &diag_last_row,
          &diag_first_col,
          &diag_last_col);
      // Calculate the position of the centre points of the two splines involved in
      // calculating the value at this point in JtJ. Note we assume that the pirst two points
      // in this diagonal are representitive of the overall diagonal
      unsigned int spl1_mid_x, spl1_mid_y, spl1_mid_z, spl2_mid_x, spl2_mid_y, spl2_mid_z;
      CBF::index_lin_to_vol(
          // Input
          diag_first_row,
          param_sz_x,
          param_sz_y,
          param_sz_z,
          // Output
          &spl1_mid_x,
          &spl1_mid_y,
          &spl1_mid_z);
      CBF::index_lin_to_vol(
          // Input
          diag_first_col,
          param_sz_x,
          param_sz_y,
          param_sz_z,
          // Output
          &spl2_mid_x,
          &spl2_mid_y,
          &spl2_mid_z);
      // Find difference in centres to calculate orientation.
      // NB this needs to be a signed operation
      auto spl_diff_mid_x = static_cast<int>(spl1_mid_x) - static_cast<int>(spl2_mid_x);
      auto spl_diff_mid_y = static_cast<int>(spl1_mid_y) - static_cast<int>(spl2_mid_y);
      auto spl_diff_mid_z = static_cast<int>(spl1_mid_z) - static_cast<int>(spl2_mid_z);
      CBF::calculate_overlap(
          // Input
          spl_diff_mid_x,
          spl_ksp_x,
          spl_order,
          // Output
          &spl1_xind_start,
          &spl1_xind_end,
          &spl2_xind_start);
      CBF::calculate_overlap(
          // Input
          spl_diff_mid_y,
          spl_ksp_y,
          spl_order,
          // Output
          &spl1_yind_start,
          &spl1_yind_end,
          &spl2_yind_start);
      CBF::calculate_overlap(
          // Input
          spl_diff_mid_z,
          spl_ksp_z,
          spl_order,
          // Output
          &spl1_zind_start,
          &spl1_zind_end,
          &spl2_zind_start);
    }
    // Wait for thread 0 here
    __syncthreads();
    // Populate shared 1D splines
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_x) - 1; ++i){
        shared_spline_x[i*blockDim.x + threadIdx.x] = spl_x[i*blockDim.x + threadIdx.x];
    }
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_y) - 1; ++i){
        shared_spline_y[i*blockDim.x + threadIdx.x] = spl_y[i*blockDim.x + threadIdx.x];
    }
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_z) - 1; ++i){
        shared_spline_z[i*blockDim.x + threadIdx.x] = spl_z[i*blockDim.x + threadIdx.x];
    }
    // Wait for all threads here
    __syncthreads();
    // Which point in the hessian is this particular thread calculating?
    int this_row = offset_into_diag;
    int this_col = offset_into_diag + diag_first_col - diag_first_row;

    // Is this a valid point?
    if (!is_valid_index(this_row, jtj_sz_diag)) return;
    else if (!is_valid_index(this_col, jtj_sz_diag)) return;
    // Calculate offset between splines
    unsigned int spl1_mid_x, spl1_mid_y, spl1_mid_z, spl2_mid_x, spl2_mid_y, spl2_mid_z;
    CBF::index_lin_to_vol(
        // Input
        this_row,
        param_sz_x,
        param_sz_y,
        param_sz_z,
        // Output
        &spl1_mid_x,
        &spl1_mid_y,
        &spl1_mid_z);
    CBF::index_lin_to_vol(
        // Input
        this_col,
        param_sz_x,
        param_sz_y,
        param_sz_z,
        // Output
        &spl2_mid_x,
        &spl2_mid_y,
        &spl2_mid_z);
    auto spl_diff_mid_x = static_cast<int>(spl1_mid_x) - static_cast<int>(spl2_mid_x);
    auto spl_diff_mid_y = static_cast<int>(spl1_mid_y) - static_cast<int>(spl2_mid_y);
    auto spl_diff_mid_z = static_cast<int>(spl1_mid_z) - static_cast<int>(spl2_mid_z);
    // !!!NB!!!NB!!!NB!!!
    // Here we take care of sparsity. We only accept those points where spl1 is "right"
    // of spl2 in all directions. I.E. spl_diff_mid_? must be > 0.
    // Additionally, if the difference is greater than the order of the spline then we are at
    // a "wrap" point, and there is no spline overlap.
    // !!!NB!!!NB!!!NB!!!
    if (!CBF::is_valid_index(spl_diff_mid_x, spl_order + 1)) return;
    else if (!CBF::is_valid_index(spl_diff_mid_y, spl_order + 1)) return;
    else if (!CBF::is_valid_index(spl_diff_mid_z, spl_order + 1)) return;
    // Use spl1 indices to calculate the corresponding indexes into the image volume
    // NOTE: These values may be negative
    int vol_xind_start =
      static_cast<int>(spl1_mid_x*spl_ksp_x) // Centre of spline in volume
      - static_cast<int>(spl_order*spl_ksp_x - 1) // Deal with spline "0" being outside volume
      + static_cast<int>(spl1_xind_start); // Deal with area of valid overlap
    int vol_yind_start =
      static_cast<int>(spl1_mid_y*spl_ksp_y) // Centre of spline in volume
      - static_cast<int>(spl_order*spl_ksp_y - 1) // Deal with spline "0" being outside volume
      + static_cast<int>(spl1_yind_start); // Deal with area of valid overlap
    int vol_zind_start =
      static_cast<int>(spl1_mid_z*spl_ksp_z) // Centre of spline in volume
      - static_cast<int>(spl_order*spl_ksp_z - 1) // Deal with spline "0" being outside volume
      + static_cast<int>(spl1_zind_start); // Deal with area of valid overlap
    // Calculate value in JtJ
    // This is done via a nested FOR loop, iterating through volume with x-direction
    // varying fastest
    int i_start = 0;
    int i_end = spl1_xind_end - spl1_xind_start;
    if (vol_xind_start < 0) i_start = -vol_xind_start;
    if (vol_xind_start + i_end >= ima_sz_x) i_end = ima_sz_x - vol_xind_start - 1;
    int j_start = 0;
    int j_end = spl1_yind_end - spl1_yind_start;
    if (vol_yind_start < 0) j_start = -vol_yind_start;
    if (vol_yind_start + j_end >= ima_sz_y) j_end = ima_sz_y - vol_yind_start - 1;
    int k_start = 0;
    int k_end = spl1_zind_end - spl1_zind_start;
    if (vol_zind_start < 0) k_start = -vol_zind_start;
    if (vol_zind_start + k_end >= ima_sz_z) k_end = ima_sz_z - vol_zind_start - 1;
    float jtj_val = 0.0;
    // The big loop
    for (int k = k_start
        ; k <= k_end
        ; ++k)
    {
      int vol_zind = vol_zind_start + static_cast<int>(k);
      for (int j = j_start
          ; j <= j_end
          ; ++j)
      {
        int vol_yind = vol_yind_start + static_cast<int>(j);
        for (int i = i_start
            ; i <= i_end
            ; ++i)
        {
          int vol_xind = vol_xind_start + static_cast<int>(i);
          // All indices are valid, therefore calculate a value
          unsigned int vol_lind = 0;
          unsigned int spl1_zind = spl1_zind_start + k;
          unsigned int spl2_zind = spl2_zind_start + k;
          unsigned int spl1_yind = spl1_yind_start + j;
          unsigned int spl2_yind = spl2_yind_start + j;
          unsigned int spl1_xind = spl1_xind_start + i;
          unsigned int spl2_xind = spl2_xind_start + i;
          CBF::index_vol_to_lin(
              // Input
              vol_xind,
              vol_yind,
              vol_zind,
              ima_sz_x,
              ima_sz_y,
              ima_sz_z,
              // Output
              &vol_lind);
          // The big calc
          jtj_val += tex1Dfetch<float>(ima,vol_lind)
              * shared_spline_x[spl1_xind]
              * shared_spline_y[spl1_yind]
              * shared_spline_z[spl1_zind]
              * shared_spline_x[spl2_xind]
              * shared_spline_y[spl2_yind]
              * shared_spline_z[spl2_zind];
        }
      }
    }
    // Calculate levels of symmetry
    unsigned int symm_1, symm_2, symm_3;
    CBF::index_lin_to_vol(
        // Input
        blockIdx.x,
        2*spl_order + 1,
        2*spl_order + 1,
        2*spl_order + 1,
        // Output
        &symm_1,
        &symm_2,
        &symm_3);
    int previous_diag_idx = blockIdx.x;
    int previous_row = this_row;
    //int previous_col = this_col;
    for (int k = 0; k <= 1; ++k){
      unsigned int symm_k;
      // Avoid redundant loops
      if (symm_3 == spl_order) ++k;
      if (k == 0) symm_k = symm_3;
      else symm_k = 2*spl_order - symm_3;
      for (int j = 0; j <= 1; ++j){
        unsigned int symm_j;
        // Avoid redundant loops
        if (symm_2 == spl_order) ++j;
        if (j == 0) symm_j = symm_2;
        else symm_j = 2*spl_order - symm_2;
        for (int i = 0; i <= 1; ++i){
          unsigned int symm_i;
          // Avoid redundant loops
          if (symm_1 == spl_order) ++i;
          if (i == 0) symm_i = symm_1;
          else symm_i = 2*spl_order - symm_1;
          unsigned int inner_diag_idx;
          CBF::index_vol_to_lin(
              // Input
              symm_i,
              symm_j,
              symm_k,
              2*spl_order + 1,
              2*spl_order + 1,
              2*spl_order + 1,
              // Output
              &inner_diag_idx);
          int diag_diff = jtj_offsets[inner_diag_idx] - jtj_offsets[previous_diag_idx];
          int inner_row = previous_row - diag_diff/2;
          previous_diag_idx = inner_diag_idx;
          previous_row = inner_row;
          // Save the value
          jtj_values[inner_diag_idx*jtj_sz_diag + inner_row] = jtj_val;
        }
      }
    }
  } // kernel_make_jtj_symmetrical

  ////////////////////////////////////////////////////////////////////////////////////////////
  /// \details This is the most important part of the code so far. We launch a single Kernel
  /// which is responsible for calculating the Hessian. The first iteration of this code was
  /// monolithic and really hard to follow, but I was paranoid about performance. This time I
  /// I am trying to do a much better job of splitting things up into useful functions. Also,
  /// I will try and give the compiler at least a passable shot at optimising by declaring
  /// everything inline!
  ///
  /// An important aspect of this kernel is the logic regarding block and thread ids. They
  /// work as follows:
  ///
  /// gridDim.x  = total number of diagonals being calculated (343 if no symmetry used)
  /// gridDim.y  = total number of chunks each diagonal is broken into
  /// blockIdx.x = number of the current diagonal in jtj, starting at 0 for the lowest
  ///              diagonal present (343 total for cubic spline, therefore main diagonal has
  ///              blockIdx.x == 171)
  /// blockIdx.y = number of the current chunk within the current diagonal
  /// blockDim.x = total number of threads launched for each block (i.e. the size of each chunk
  ///              within each diagonal)
  ///
  /// \param ima_sz_x size of image in x-direction
  /// \param ima_sz_y size of image in y-direction
  /// \param ima_sz_z size of image in z-direction
  /// \param ima pre-multiplied image volume
  /// \param spl_x_1 1D spline kernel in x-direction
  /// \param spl_y_1 1D spline kernel in y-direction
  /// \param spl_z_1 1D spline kernel in z-direction
  /// \param spl_x_2 1D spline kernel in x-direction
  /// \param spl_y_2 1D spline kernel in y-direction
  /// \param spl_z_2 1D spline kernel in z-direction
  /// \param spl_ksp_x_1 knot spacing of spline in x-direction
  /// \param spl_ksp_y_1 knot spacing of spline in y-direction
  /// \param spl_ksp_z_1 knot spacing of spline in z-direction
  /// \param spl_ksp_x_2 knot spacing of spline in x-direction
  /// \param spl_ksp_y_2 knot spacing of spline in y-direction
  /// \param spl_ksp_z_2 knot spacing of spline in z-direction
  /// \param param_sz_x_1 size of parameter space in x-direction
  /// \param param_sz_y_1 size of parameter space in y-direction
  /// \param param_sz_z_1 size of parameter space in z-direction
  /// \param param_sz_x_2 size of parameter space in x-direction
  /// \param param_sz_y_2 size of parameter space in y-direction
  /// \param param_sz_z_2 size of parameter space in z-direction
  /// \param jtj_offsets offsets of sparse diagonal representation of jtj
  ///
  /// \param jtj_values linearise values of sparse diagonal representation of jtj
  ////////////////////////////////////////////////////////////////////////////////////////////
  __global__ void kernel_make_jtj_non_symmetrical(
      // Input
      unsigned int ima_sz_x,
      unsigned int ima_sz_y,
      unsigned int ima_sz_z,
      hipTextureObject_t ima,
      const float* __restrict__ spl_x_1,
      const float* __restrict__ spl_y_1,
      const float* __restrict__ spl_z_1,
      const float* __restrict__ spl_x_2,
      const float* __restrict__ spl_y_2,
      const float* __restrict__ spl_z_2,
      unsigned int spl_ksp_x_1,
      unsigned int spl_ksp_y_1,
      unsigned int spl_ksp_z_1,
      unsigned int spl_ksp_x_2,
      unsigned int spl_ksp_y_2,
      unsigned int spl_ksp_z_2,
      unsigned int param_sz_x_1,
      unsigned int param_sz_y_1,
      unsigned int param_sz_z_1,
      unsigned int param_sz_x_2,
      unsigned int param_sz_y_2,
      unsigned int param_sz_z_2,
      const int* __restrict__ jtj_offsets,
      // Output
      float* __restrict__ jtj_values)
  {
    __shared__ unsigned int diag_first_row;
    __shared__ unsigned int diag_last_row;
    __shared__ unsigned int diag_first_col;
    __shared__ unsigned int diag_last_col;
    extern __shared__ float all_splines[];
    const int spl_order = 3; // Might replace this with a parameter later
    const auto jtj_sz_diag = param_sz_x_1 * param_sz_y_1 * param_sz_z_1;
    const auto jtj_sz_row = jtj_sz_diag;
    const auto jtj_sz_col = param_sz_x_2 * param_sz_y_2 * param_sz_z_2;
    const auto offset_into_diag = blockIdx.y*blockDim.x + threadIdx.x;
    float *shared_spline_x_1 = all_splines;
    float *shared_spline_y_1 = &shared_spline_x_1[spl_ksp_x_1 * (spl_order + 1) - 1];
    float *shared_spline_z_1 = &shared_spline_y_1[spl_ksp_y_1 * (spl_order + 1) - 1];
    float *shared_spline_x_2 = &shared_spline_z_1[spl_ksp_z_1 * (spl_order + 1) - 1];
    float *shared_spline_y_2 = &shared_spline_x_2[spl_ksp_x_2 * (spl_order + 1) - 1];
    float *shared_spline_z_2 = &shared_spline_y_2[spl_ksp_y_2 * (spl_order + 1) - 1];

    // Calculate the overlapping regions of each spline. This only needs to be calculated
    // once per block, as all threads within a block are characterised by the same type of
    // overlap (except when the splines reach the end of the image in any dimension)
    // NB!!! Calculating the overlap once per diagonal only works if the diagonal starts
    // with a valid overlap!!! I.E. this will not work when symmetry is does not hold!!!
    if (threadIdx.x == 0){
      // Identify the first valid (row,col) pair for this particular block
      auto this_offset = jtj_offsets[blockIdx.x];
      CBF::calculate_diagonal_range(
          // Input
          this_offset,
          jtj_sz_diag,
          jtj_sz_diag,
          // Output
          &diag_first_row,
          &diag_last_row,
          &diag_first_col,
          &diag_last_col);
      // Calculate the position of the centre points of the two splines involved in
      // calculating the value at this point in JtJ. Note we assume that the pirst two points
      // in this diagonal are representitive of the overall diagonal
      unsigned int spl1_mid_x, spl1_mid_y, spl1_mid_z, spl2_mid_x, spl2_mid_y, spl2_mid_z;
      CBF::index_lin_to_vol(
          // Input
          diag_first_row,
          param_sz_x_1,
          param_sz_y_1,
          param_sz_z_1,
          // Output
          &spl1_mid_x,
          &spl1_mid_y,
          &spl1_mid_z);
      CBF::index_lin_to_vol(
          // Input
          diag_first_col,
          param_sz_x_2,
          param_sz_y_2,
          param_sz_z_2,
          // Output
          &spl2_mid_x,
          &spl2_mid_y,
          &spl2_mid_z);
    }
    // Wait for thread 0 here
    __syncthreads();
    // Populate shared 1D splines
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_x_1) - 1; ++i){
        shared_spline_x_1[i*blockDim.x + threadIdx.x] = spl_x_1[i*blockDim.x + threadIdx.x];
    }
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_y_1) - 1; ++i){
        shared_spline_y_1[i*blockDim.x + threadIdx.x] = spl_y_1[i*blockDim.x + threadIdx.x];
    }
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_z_1) - 1; ++i){
        shared_spline_z_1[i*blockDim.x + threadIdx.x] = spl_z_1[i*blockDim.x + threadIdx.x];
    }
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_x_2) - 1; ++i){
        shared_spline_x_2[i*blockDim.x + threadIdx.x] = spl_x_2[i*blockDim.x + threadIdx.x];
    }
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_y_2) - 1; ++i){
        shared_spline_y_2[i*blockDim.x + threadIdx.x] = spl_y_2[i*blockDim.x + threadIdx.x];
    }
    for (int i = 0; i*blockDim.x + threadIdx.x < ((spl_order + 1) * spl_ksp_z_2) - 1; ++i){
        shared_spline_z_2[i*blockDim.x + threadIdx.x] = spl_z_2[i*blockDim.x + threadIdx.x];
    }
    // Wait for all threads here
    __syncthreads();
    // Which point in the hessian is this particular thread calculating?
    int this_row = offset_into_diag;
    int this_col = offset_into_diag + diag_first_col - static_cast<int>(diag_first_row);
    // Is this a valid point?
    if (!is_valid_index(this_row, jtj_sz_row)) return;
    else if (!is_valid_index(this_col, jtj_sz_col)) return;
    // Calculate offset between splines
    unsigned int spl1_mid_x, spl1_mid_y, spl1_mid_z, spl2_mid_x, spl2_mid_y, spl2_mid_z;
    CBF::index_lin_to_vol(
        // Input
        this_row,
        param_sz_x_1,
        param_sz_y_1,
        param_sz_z_1,
        // Output
        &spl1_mid_x,
        &spl1_mid_y,
        &spl1_mid_z);
    CBF::index_lin_to_vol(
        // Input
        this_col,
        param_sz_x_2,
        param_sz_y_2,
        param_sz_z_2,
        // Output
        &spl2_mid_x,
        &spl2_mid_y,
        &spl2_mid_z);
    // Find difference in centres to calculate orientation.
    // NB this needs to be a signed operation
    auto spl_diff_mid_x = static_cast<int>(spl1_mid_x) - static_cast<int>(spl2_mid_x);
    auto spl_diff_mid_y = static_cast<int>(spl1_mid_y) - static_cast<int>(spl2_mid_y);
    auto spl_diff_mid_z = static_cast<int>(spl1_mid_z) - static_cast<int>(spl2_mid_z);
    // !!!NB!!!NB!!!NB!!!
    // Here we take care of sparsity. We only accept those points where spl1 is "right"
    // of spl2 in all directions. I.E. spl_diff_mid_? must be > 0.
    // Additionally, if the difference is greater than the order of the spline then we are at
    // a "wrap" point, and there is no spline overlap.
    // !!!NB!!!NB!!!NB!!!
    if (!CBF::is_valid_index(spl_diff_mid_x + spl_order, 2*spl_order + 1)) return;
    else if (!CBF::is_valid_index(spl_diff_mid_y + spl_order, 2*spl_order + 1)) return;
    else if (!CBF::is_valid_index(spl_diff_mid_z + spl_order, 2*spl_order + 1)) return;
    // We actually only need to calculate the valid spline indices once per diagonal
    // This could potentially be done outside of the kernel in fact, but would require a
    // a fairly major reworking of the logic involved, but still potentially worth it.
    unsigned int spl1_xind_start, spl1_xind_end, spl2_xind_start;
    unsigned int spl1_yind_start, spl1_yind_end, spl2_yind_start;
    unsigned int spl1_zind_start, spl1_zind_end, spl2_zind_start;
    // If this is a real point, calculate the overlap
    CBF::calculate_overlap(
        // Input
        spl_diff_mid_x,
        spl_ksp_x_1,
        spl_order,
        // Output
        &spl1_xind_start,
        &spl1_xind_end,
        &spl2_xind_start);
    CBF::calculate_overlap(
        // Input
        spl_diff_mid_y,
        spl_ksp_y_1,
        spl_order,
        // Output
        &spl1_yind_start,
        &spl1_yind_end,
        &spl2_yind_start);
    CBF::calculate_overlap(
        // Input
        spl_diff_mid_z,
        spl_ksp_z_1,
        spl_order,
        // Output
        &spl1_zind_start,
        &spl1_zind_end,
        &spl2_zind_start);
    // Use spl1 indices to calculate the corresponding indexes into the image volume
    // NOTE: These values may be negative
    int vol_xind_start =
      static_cast<int>(spl1_mid_x*spl_ksp_x_1) // Centre of spline in volume
      - static_cast<int>(spl_order*spl_ksp_x_1 - 1) // Deal with spline "0" being outside volume
      + static_cast<int>(spl1_xind_start); // Deal with area of valid overlap
    int vol_yind_start =
      static_cast<int>(spl1_mid_y*spl_ksp_y_1) // Centre of spline in volume
      - static_cast<int>(spl_order*spl_ksp_y_1 - 1) // Deal with spline "0" being outside volume
      + static_cast<int>(spl1_yind_start); // Deal with area of valid overlap
    int vol_zind_start =
      static_cast<int>(spl1_mid_z*spl_ksp_z_1) // Centre of spline in volume
      - static_cast<int>(spl_order*spl_ksp_z_1 - 1) // Deal with spline "0" being outside volume
      + static_cast<int>(spl1_zind_start); // Deal with area of valid overlap
    // Calculate value in JtJ
    // This is done via a nested FOR loop, iterating through volume with x-direction
    // varying fastest
    int i_start = 0;
    int i_end = spl1_xind_end - spl1_xind_start;
    if (vol_xind_start < 0) i_start = -vol_xind_start;
    if (vol_xind_start + i_end >= ima_sz_x) i_end = ima_sz_x - vol_xind_start - 1;
    int j_start = 0;
    int j_end = spl1_yind_end - spl1_yind_start;
    if (vol_yind_start < 0) j_start = -vol_yind_start;
    if (vol_yind_start + j_end >= ima_sz_y) j_end = ima_sz_y - vol_yind_start - 1;
    int k_start = 0;
    int k_end = spl1_zind_end - spl1_zind_start;
    if (vol_zind_start < 0) k_start = -vol_zind_start;
    if (vol_zind_start + k_end >= ima_sz_z) k_end = ima_sz_z - vol_zind_start - 1;
    float jtj_val = 0.0;
    // The big loop
    for (int k = k_start
        ; k <= k_end
        ; ++k)
    {
      int vol_zind = vol_zind_start + k;
      for (int j = j_start
          ; j <= j_end
          ; ++j)
      {
        int vol_yind = vol_yind_start + j;
        for (int i = i_start
            ; i <= i_end
            ; ++i)
        {
          int vol_xind = vol_xind_start + i;
          // All indices are valid, therefore calculate a value
          unsigned int vol_lind = 0;
          unsigned int spl1_zind = spl1_zind_start + k;
          unsigned int spl2_zind = spl2_zind_start + k;
          unsigned int spl1_yind = spl1_yind_start + j;
          unsigned int spl2_yind = spl2_yind_start + j;
          unsigned int spl1_xind = spl1_xind_start + i;
          unsigned int spl2_xind = spl2_xind_start + i;
          CBF::index_vol_to_lin(
              // Input
              vol_xind,
              vol_yind,
              vol_zind,
              ima_sz_x,
              ima_sz_y,
              ima_sz_z,
              // Output
              &vol_lind);
          // The big calc
          jtj_val += tex1Dfetch<float>(ima,vol_lind)
              * shared_spline_x_1[spl1_xind]
              * shared_spline_y_1[spl1_yind]
              * shared_spline_z_1[spl1_zind]
              * shared_spline_x_2[spl2_xind]
              * shared_spline_y_2[spl2_yind]
              * shared_spline_z_2[spl2_zind];
        }
      }
    }
    // Save value
    int diag_idx = blockIdx.x;
    jtj_values[diag_idx*jtj_sz_diag + this_row] = jtj_val;
  } // kernel_make_jtj_symmetrical
} // CBFKernels definitions

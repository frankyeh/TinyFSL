#include "hip/hip_runtime.h"
/*! \file LSResampler.cu
    \brief Contains definition of CUDA implementation of a class for least-squares resampling of pairs of images

    \author Jesper Andersson
    \version 1.0b, August, 2013.
*/
//
// LSResampler.cu
//
// Jesper Andersson, FMRIB Image Analysis Group
//
// Copyright (C) 2013 University of Oxford
//

// Because of a bug in cuda_fp16.hpp, that gets included by hipblas.h, it has to
// be included before any include files that set up anything related to the std-lib.
// If not, there will be an ambiguity in cuda_fp16.hpp about wether to use the
// old-style C isinf or the new (since C++11) std::isinf.
#include "hipblas.h"

#include <cstdlib>
#include <string>
#include <vector>
#include <cmath>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#pragma push
#pragma diag_suppress = code_is_unreachable // Supress warnings from armawrap
#pragma diag_suppress = expr_has_no_effect  // Supress warnings from boost
#include "newimage/newimageall.h"
#pragma pop
#include "topup/topup_file_io.h"
#include "topup/displacement_vector.h"
#include "EddyHelperClasses.h"
#include "EddyUtils.h"
#include "ECScanClasses.h"
#include "LSResampler.h"
#include "EddyCudaHelperFunctions.h"
#include "EddyKernels.h"
#include "EddyMatrixKernels.h"
#include "CudaVolume.h"
#include "EddyInternalGpuUtils.h"
#include "EddyGpuUtils.h"

namespace EDDY {

class LSResamplerImpl
{
public:
  LSResamplerImpl(const EDDY::ECScan&                             s1,
		  const EDDY::ECScan&                             s2,
		  std::shared_ptr<const NEWIMAGE::volume<float> > hzfield,
		  double                                          lambda);
  const NEWIMAGE::volume<float>& GetResampledVolume() const EddyTry { return(_rvol); } EddyCatch
  const NEWIMAGE::volume<float>& GetMask() const EddyTry { return(_omask); } EddyCatch
private:
  NEWIMAGE::volume<float>  _rvol;  // Resampled volume
  NEWIMAGE::volume<float>  _omask; // Mask indicating valid voxels in _rvol
  // These values have all been arrived at by trial and error
  static const int         _threads_per_block_QR = 128;
  static const int         _threads_per_block_Solve = 128;
  static const int         _threads_per_block_Kty = 128;
  static const dim3        _threads_per_block_KtK;

  template<typename T>
  T sqr(const T& v) const { return(v*v); }

  unsigned int make_k_matrices(// Input
			       const NEWIMAGE::volume4D<float>& field1,  // Field for first image
			       const NEWIMAGE::volume4D<float>& field2,  // Field for 2nd image
			       const NEWIMAGE::volume<float>&   mask,    // Mask saying where field is valid
			       unsigned int                     sl,      // Current slice
			       bool                             pex,     // True of phase encode in x
			       // Output
			       thrust::device_vector<float>&    kmats,   // All valid K-matrices for the slice. Column first (NEWMAT convention).
			       std::vector<bool>&               isok);   // Indicates if a row/column has a valid matrix

  void solve_for_y_hat(// Input
		       const thrust::device_vector<float>& ktkmats, // KtK matrices for a slice. Row first (i.e. NOT NEWMAT convention)
		       const thrust::device_vector<float>& ktyvecs, // Kty vectors for a slice
		       unsigned int                        n,       // Size of KtK (nxn)
		       unsigned int                        nmat,    // Number of valid matrices for the slice
		       bool                                sync,    // If true, syncs after submitting kernel
		       // Output
		       thrust::device_vector<float>&       yhat);   // Returns inv(KtK)*Kty for all matrices in slice

  void make_kty_vectors(// Input
			const thrust::device_vector<float>& kmats, // All valid K-matrices for a slice. Column first (NEWMAT convention).
			const thrust::device_vector<float>& yvecs, // All valid data-vectors for a slice
			unsigned int                        m,     // Number of rows of K
			unsigned int                        n,     // Number of columns of K
			unsigned int                        nmat,  // Number of valid matrices
			bool                                sync,  // If true, syncs after submitting kernel
			// Output
			thrust::device_vector<float>&       kty);  // Kt*y for all valid matrices

  void make_ktk_matrices(// Input
			 const thrust::device_vector<float>& kmats, // All valid K-matrices for one slice. Column first (NEWMAT convention)
			 unsigned int                        m,     // Number of rows of K
			 unsigned int                        n,     // Number of columns of K
			 unsigned int                        nmat,  // Number of matrices
			 const thrust::device_vector<float>& sts,   // Regularisation matrix to add to Kt*K
			 bool                                sync,  // If true, syncs after submitting kernel
			 // Output
			 thrust::device_vector<float>&       ktk);  // Kt*K Row first (i.e. NOT NEWMAT convention)

  void transfer_y_hat_vectors(// Input
			      const thrust::device_vector<float>& yhatvecs, // All valid solution vectors for one slice
			      const std::vector<bool>&            isok,     // Vector that indicate if a row/column has a valid solution
			      unsigned int                        sl,       // Slice number
			      bool                                pex,      // True if phase-encoding along the first dimension
			      // Output
			      NEWIMAGE::volume<float>&            ima,      // Image to put solution vectors into
			      NEWIMAGE::volume<float>&            mask);    // Mask set to zero for all invalid columns/rows

  void transfer_y_vectors(// Input
			  const NEWIMAGE::volume<float>& ima1,   // First image
			  const NEWIMAGE::volume<float>& ima2,   // Second image. ima1 and ima2 must have phase-encoding in same dimension
			  const std::vector<bool>&       isok,   // Vector that indicate if a row/column has a valid solution
			  unsigned int                   sl,     // Slice number
			  bool                           pex,    // True if phase-encoding along the first dimension
			  // Output
			  thrust::device_vector<float>&  yvecs); // Vector on the GPU that vectors are transferred to

  void copy_row_first_to_matrix(const float      *rf_fptr,
				NEWMAT::Matrix&  M);

  void copy_to_matrix(const float      *fptr,  // Source-vector whos length should be M.Nrows()*M.Ncols()
		      NEWMAT::Matrix&  M);     // Destination NEWMAT matrix/vector

  void copy_matrix(const NEWMAT::Matrix& M,      // Source NEWMAT matric/vector
		   float                 *fptr); // Destination vector whos length should be M.Nrows()*M.Ncols()

  void get_sts(unsigned int                  sz,      // Size of StS (StS is sz \times sz)
	       double                        lambda,  // Scalar/weight to multiply StS by
	       thrust::device_vector<float>& dStS);   // lambda*StS on the GPU

  void dump_matrices(const std::string&                  fname,      // File name for dump
		     const thrust::device_vector<float>& mats,       // Matrices (on device) to dump
		     unsigned int                        m,          // No of rows for one matrix
		     unsigned int                        n,          // No of columns for one matrix
		     unsigned int                        nmat,       // No of matrices
		     bool                                row_first); // True if row index varies fastest (opposite to NEWMAT)
};
const dim3 LSResamplerImpl::_threads_per_block_KtK = dim3(16,16);

LSResampler::LSResampler(const EDDY::ECScan&                             s1,
			 const EDDY::ECScan&                             s2,
			 std::shared_ptr<const NEWIMAGE::volume<float> > hzfield,
			 double                                          lambda) EddyTry
{
  _pimpl = new LSResamplerImpl(s1,s2,hzfield,lambda);
} EddyCatch

LSResampler::~LSResampler()
{
  delete _pimpl;
}

const NEWIMAGE::volume<float>& LSResampler::GetResampledVolume() const EddyTry
{
  return(_pimpl->GetResampledVolume());
} EddyCatch

const NEWIMAGE::volume<float>& LSResampler::GetMask() const EddyTry
{
  return(_pimpl->GetMask());
} EddyCatch

/****************************************************************//**
*
* Constructs an LSResamplerImpl object.
* All the work for resampling a pair of scans into a single volume
* is carried out inside this constructor. After the object has been
* constructed one can immediately obtain the resampled volume through
* a call to GetResampledVolume.
* \param s1 One of a pair of scans with parallel phase-encodings
* \param s2 The second of a pair of scans with parallel phase-encodings
* \param hzfield Field in Hz in model space
* \param lambda Weight of regularisation
*
********************************************************************/
LSResamplerImpl::LSResamplerImpl(const EDDY::ECScan&                             s1,
				 const EDDY::ECScan&                             s2,
				 std::shared_ptr<const NEWIMAGE::volume<float> > hzfield,
				 double                                          lambda) EddyTry
{
  if (!EddyUtils::AreMatchingPair(s1,s2)) throw EddyException("LSResampler::LSResampler:: Mismatched pair");
  EddyCudaHelperFunctions::InitGpu();
  // Resample both images using rigid body parameters
  NEWIMAGE::volume<float> ima1;
  NEWIMAGE::volume<float> mask;
  EddyGpuUtils::GetMotionCorrectedScan(s1,false,ima1,&mask);
  NEWIMAGE::volume<float> ima2;
  NEWIMAGE::volume<float> mask2;
  EddyGpuUtils::GetMotionCorrectedScan(s2,false,ima2,&mask2);
  mask *= mask2;

  _omask.reinitialize(mask.xsize(),mask.ysize(),mask.zsize());
  _omask = 1.0;
  _rvol.reinitialize(ima1.xsize(),ima1.ysize(),ima1.zsize());
  NEWIMAGE::copybasicproperties(ima1,_rvol);
  // Get fields
  NEWIMAGE::volume4D<float> field1 = s1.FieldForScanToModelTransform(hzfield); // In mm
  NEWIMAGE::volume4D<float> field2 = s2.FieldForScanToModelTransform(hzfield); // In mm

  // Check what direction phase-encode is in and get relevant sizes
  bool pex = false;
  unsigned int matsz = field1[0].ysize();
  if (s1.GetAcqPara().PhaseEncodeVector()(1)) { pex = true; matsz = field1[0].xsize(); }
  unsigned int nK = matsz; unsigned int mK = 2*nK;
  unsigned int nmat_per_sl = (pex) ? ima1.ysize() : ima1.xsize();

  // Allocate memory for storing matrices on the GPU
  thrust::device_vector<float> gpu_K_matrices(nmat_per_sl*2*sqr(matsz),0.0);
  thrust::device_vector<float> gpu_KtK_matrices(nmat_per_sl*sqr(matsz),0.0);
  thrust::device_vector<float> gpu_StS(sqr(matsz),0.0);
  thrust::device_vector<float> gpu_y_vectors(nmat_per_sl*2*matsz,0.0);
  thrust::device_vector<float> gpu_Kty_vectors(nmat_per_sl*matsz,0.0);
  thrust::device_vector<float> gpu_solution_vectors(nmat_per_sl*matsz,0.0);
  // Allocate some additional stuff
  std::vector<bool> isok(nmat_per_sl);
  // Get regularisation matrix once and for all
  get_sts(matsz,lambda,gpu_StS);

  for (int k=0; k<ima1.zsize(); k++) {
    // Make all K-matrices for this slice. This is a CPU-only job (+transfer to GPU)
    unsigned int nmat = make_k_matrices(field1,field2,mask,k,pex,gpu_K_matrices,isok);
    // Transfer all y-vectors for this slice.
    transfer_y_vectors(ima1,ima2,isok,k,pex,gpu_y_vectors);
    // Multiply to make KtK matrices. This is a GPU job.
    make_ktk_matrices(gpu_K_matrices,mK,nK,nmat,gpu_StS,true,gpu_KtK_matrices);
    // Multiply y-vectors with K'. This is a GPU job.
    make_kty_vectors(gpu_K_matrices,gpu_y_vectors,mK,nK,nmat,true,gpu_Kty_vectors);
    // Solve for y_hat. This is a GPU job.
    solve_for_y_hat(gpu_KtK_matrices,gpu_Kty_vectors,nK,nmat,true,gpu_solution_vectors);
    // Insert solutions into image
    transfer_y_hat_vectors(gpu_solution_vectors,isok,k,pex,_rvol,_omask);
  }

  return;
} EddyCatch

unsigned int LSResamplerImpl::make_k_matrices(// Input
					      const NEWIMAGE::volume4D<float>& field1,  // Field for first image
					      const NEWIMAGE::volume4D<float>& field2,  // Field for 2nd image
					      const NEWIMAGE::volume<float>&   mask,    // Mask saying where field is valid
					      unsigned int                     sl,      // Current slice
					      bool                             pex,     // True of phase encode in x
					      // Output
					      thrust::device_vector<float>&    kmats,   // All valid K-matrices for the whole slice
					      std::vector<bool>&               isok) EddyTry // Indicates if a row/column has a valid matrix
{
  unsigned int sz = (pex) ? field1.xsize() : field1.ysize();
  unsigned int nmat_per_sl = (pex) ? field1.ysize() : field1.xsize();
  unsigned int matsize = 2*sz*sz;
  TOPUP::DispVec dv1(sz), dv2(sz);
  thrust::host_vector<float> host_kmats(nmat_per_sl*matsize,0.0);

  double sf1, sf2; // Scale factors mm->voxels
  if (pex) { sf1 = 1.0/field1.xdim(); sf2 = 1.0/field2.xdim(); }
  else { sf1 = 1.0/field1.ydim(); sf2 = 1.0/field2.ydim(); }

  float *kptr = thrust::raw_pointer_cast(host_kmats.data());
  unsigned int nvalid = 0; // Number of valid matrices
  for (int i=0; i<nmat_per_sl; i++) {
    bool row_col_is_ok = true;
    if (pex) {
      if (!dv1.RowIsAlright(mask,sl,i)) row_col_is_ok = false;
      else {
	dv1.SetFromRow(field1[0],sl,i);
	dv2.SetFromRow(field2[0],sl,i);
      }
    }
    else {
      if (!dv1.ColumnIsAlright(mask,sl,i)) row_col_is_ok = false;
      else {
	dv1.SetFromColumn(field1[1],sl,i);
	dv2.SetFromColumn(field2[1],sl,i);
      }
    }
    if (row_col_is_ok) {
      isok[i] = true;
      NEWMAT::Matrix K = dv1.GetK_Matrix(sf1) & dv2.GetK_Matrix(sf2);
      copy_matrix(K,kptr);
      kptr+=matsize;
      nvalid++;
    }
    else isok[i] = false;
  }
  kmats = host_kmats; // Host->GPU
  return(nvalid);
} EddyCatch

void LSResamplerImpl::solve_for_y_hat(// Input
				      const thrust::device_vector<float>& ktkmats, // KtK matrices for a slice. Row first (i.e. NOT NEWMAT convention)
				      const thrust::device_vector<float>& ktyvecs, // Kty vectors for a slice
				      unsigned int                        n,       // Size of KtK (nxn)
				      unsigned int                        nmat,    // Number of valid matrices for slice
				      bool                                sync,    // If true syncs after submitting kernel
				      // Output
				      thrust::device_vector<float>&       yhat) EddyTry // Returns inv(KtK)*Kty for all matrices in slice
{
  if (nmat) {
    // Allocate memory for Q and R matrices for QR decomposition
    thrust::device_vector<float> Qt(nmat*n*n,0.0);
    thrust::device_vector<float> R(nmat*n*n,0.0);
    // Dynamically allocated shared memory, per block (matrix)
    size_t sh_mem_sz = 2*n*sizeof(float);
    int tpb = _threads_per_block_QR;
    EddyMatrixKernels::QR<<<nmat,tpb,sh_mem_sz>>>(thrust::raw_pointer_cast(ktkmats.data()),n,n,nmat,
						  thrust::raw_pointer_cast(Qt.data()),
						  thrust::raw_pointer_cast(R.data()));
    if (sync) EddyCudaHelperFunctions::CudaSync("QR_Kernels::QR");
    tpb = _threads_per_block_Solve;
    EddyMatrixKernels::Solve<<<nmat,tpb>>>(thrust::raw_pointer_cast(Qt.data()),
					   thrust::raw_pointer_cast(R.data()),
					   thrust::raw_pointer_cast(ktyvecs.data()),n,n,nmat,
					   thrust::raw_pointer_cast(yhat.data()));
    if (sync) EddyCudaHelperFunctions::CudaSync("QR_Kernels::Solve");
  }
  else {
    thrust::fill(yhat.begin(),yhat.end(),static_cast<float>(0.0));
  }
  return;
} EddyCatch

void LSResamplerImpl::dump_matrices(const std::string&                  fname,      // File name for dump
				    const thrust::device_vector<float>& mats,       // Matrices (on device) to dump
				    unsigned int                        m,          // No of rows for one matrix
				    unsigned int                        n,          // No of columns for one matrix
				    unsigned int                        nmat,       // No of matrices
				    bool                                row_first) EddyTry // True if row index varies fastest
{
  thrust::host_vector<float> h_mats = mats; // Transfer to CPU
  NEWMAT::Matrix big_m(m,n);
  NEWMAT::Matrix little_m(m,n);
  if (row_first) copy_row_first_to_matrix(thrust::raw_pointer_cast(h_mats.data()),big_m);
  else copy_to_matrix(thrust::raw_pointer_cast(h_mats.data()),big_m);
  for (unsigned int mat=1; mat<nmat; mat++) {
    if (row_first) copy_row_first_to_matrix(thrust::raw_pointer_cast(h_mats.data())+mat*m*n,little_m);
    else copy_to_matrix(thrust::raw_pointer_cast(h_mats.data())+mat*m*n,little_m);
    big_m &= little_m;
  }
  MISCMATHS::write_ascii_matrix(fname,big_m);
} EddyCatch

void LSResamplerImpl::make_kty_vectors(// Input
				       const thrust::device_vector<float>& kmats, // All valid K-matrices for a slice. Column first (NEWMAT convention).
				       const thrust::device_vector<float>& yvecs, // All valid data-vectors for a slice
				       unsigned int                        m,     // Number of rows of K
				       unsigned int                        n,     // Number of columns of K
				       unsigned int                        nmat,  // Number of valid matrices
				       bool                                sync,  // If true, syncs after submitting kernel
				       // Output
				       thrust::device_vector<float>&       kty) EddyTry // Kt*y for all valid matrices
{
  if (nmat) {
    int tpb = _threads_per_block_Kty;
    EddyMatrixKernels::Kty<<<nmat,tpb>>>(thrust::raw_pointer_cast(kmats.data()),
					 thrust::raw_pointer_cast(yvecs.data()),m,n,nmat,
					 thrust::raw_pointer_cast(kty.data()));
    if (sync) EddyCudaHelperFunctions::CudaSync("KtK_Kernels::Kty");
  }
  return;
} EddyCatch

void LSResamplerImpl::make_ktk_matrices(// Input
					const thrust::device_vector<float>& kmats, // All valid K-matrices for one slice. Column first (NEWMAT convention)
					unsigned int                        m,     // Number of rows of K
					unsigned int                        n,     // Number of columns of K
					unsigned int                        nmat,  // Number of matrices
					const thrust::device_vector<float>& sts,   // Regularisation matrix to add to Kt*K
					bool                                sync,  // If true, syncs after submitting kernel
					// Output
					thrust::device_vector<float>&       ktk) EddyTry // Kt*K Row first (i.e. NOT NEWMAT convention)
{
  if (nmat) {
    dim3 block = _threads_per_block_KtK;
    EddyMatrixKernels::KtK<<<nmat,block>>>(thrust::raw_pointer_cast(kmats.data()),m,n,nmat,
					   thrust::raw_pointer_cast(sts.data()),1.0,false,
					   thrust::raw_pointer_cast(ktk.data()));
    if (sync) EddyCudaHelperFunctions::CudaSync("KtK_Kernels::KtK");
  }
  return;
} EddyCatch

void LSResamplerImpl::transfer_y_hat_vectors(// Input
					     const thrust::device_vector<float>& yhatvecs, // All valid solution vectors for one slice
					     const std::vector<bool>&            isok,     // Vector that indicate if a row/column has a valid solution
					     unsigned int                        sl,       // The slice
					     bool                                pex,      // True if phase-encoding along the first dimension
					     // Output
					     NEWIMAGE::volume<float>&            ima,      // Image to put solution vectors into
					     NEWIMAGE::volume<float>&            mask) EddyTry // Mask set to zero for all invalid columns/rows
{
  thrust::host_vector<float> host_yhatvecs = yhatvecs; // GPU->host
  unsigned int vecsize = (pex) ? ima.xsize() : ima.ysize();
  unsigned int nmat = (pex) ? ima.ysize() : ima.xsize();
  NEWMAT::ColumnVector yhat(vecsize);
  NEWMAT::ColumnVector zeros(vecsize); zeros=0.0;
  float *yhatptr = thrust::raw_pointer_cast(host_yhatvecs.data());
  for (unsigned int i=0; i<nmat; i++) {
    if (isok[i]) {
      copy_to_matrix(yhatptr,yhat);
      if (pex) ima.SetRow(i,sl,yhat); else ima.SetColumn(i,sl,yhat);
      yhatptr+=vecsize;
    }
    else {
      if (pex) { ima.SetRow(i,sl,zeros); mask.SetRow(i,sl,zeros); }
      else { ima.SetColumn(i,sl,zeros); mask.SetColumn(i,sl,zeros); }
    }
  }
} EddyCatch

void LSResamplerImpl::transfer_y_vectors(// Input
					 const NEWIMAGE::volume<float>& ima1,   // First image
					 const NEWIMAGE::volume<float>& ima2,   // Second image. ima1 and ima2 must have phase-encoding in same dimension
					 const std::vector<bool>&       isok,   // Vector that indicate if a row/column has a valid solution
					 unsigned int                   sl,     // Slice number
					 bool                           pex,    // True if phase-encoding along the first dimension
					 // Output
					 thrust::device_vector<float>&  yvecs) EddyTry // Vector on the GPU that vectors are transferred to
{
  unsigned int nvecs = (pex) ? ima1.ysize() : ima1.xsize();
  unsigned int vecsize = (pex) ? ima1.xsize() : ima1.ysize();
  thrust::host_vector<float> hyvecs(2*nvecs*vecsize,0.0);
  float *vecptr = thrust::raw_pointer_cast(hyvecs.data());
  NEWMAT::ColumnVector y(2*vecsize);
  for (int ij=0; ij<nvecs; ij++) {
    if (isok[ij]) {
      if (pex) y = ima1.ExtractRow(ij,sl) & ima2.ExtractRow(ij,sl);
      else y = ima1.ExtractColumn(ij,sl) & ima2.ExtractColumn(ij,sl);
      copy_matrix(y,vecptr);
      vecptr+=2*vecsize;
    }
  }
  yvecs = hyvecs; // Host->GPU
  return;
} EddyCatch

void LSResamplerImpl::copy_row_first_to_matrix(const float      *rf_fptr,
					       NEWMAT::Matrix&  M) EddyTry
{
  for (unsigned int r=0; r<M.Nrows(); r++) {
    for (unsigned int c=0; c<M.Ncols(); c++) {
      M(r+1,c+1) = rf_fptr[c*M.Nrows()+r];
    }
  }
} EddyCatch

void LSResamplerImpl::copy_to_matrix(const float      *fptr,     // Source-vector whos length should be M.Nrows()*M.Ncols()
				     NEWMAT::Matrix&  M) EddyTry // Destination NEWMAT matrix/vector
{
  for (unsigned int r=0; r<M.Nrows(); r++) {
    for (unsigned int c=0; c<M.Ncols(); c++) {
      M(r+1,c+1) = fptr[r*M.Ncols()+c];
    }
  }
  return;
  // Old code from before Armadillo
  /*
  double *dptr = M.Store();
  unsigned int sz = M.Nrows()*M.Ncols();
  for (int i=0; i<sz; i++, fptr++, dptr++) *dptr = static_cast<double>(*fptr);
  */
} EddyCatch

void LSResamplerImpl::copy_matrix(const NEWMAT::Matrix& M,             // Source NEWMAT matric/vector
				  float                 *fptr) EddyTry // Destination vector whos length should be M.Nrows()*M.Ncols()
{
  for (unsigned int r=0; r<M.Nrows(); r++) {
    for (unsigned int c=0; c<M.Ncols(); c++) {
      fptr[r*M.Ncols()+c] = static_cast<float>(M(r+1,c+1));
    }
  }
  return;
  // Old code from before Armadillo
  /*
  double *dptr = M.Store();
  unsigned int sz = M.Nrows()*M.Ncols();
  for (int i=0; i<sz; i++, fptr++, dptr++) *fptr = static_cast<float>(*dptr);
  */
} EddyCatch

void LSResamplerImpl::get_sts(unsigned int                  sz,
			      double                        lambda,
			      thrust::device_vector<float>& dStS) EddyTry
{
  TOPUP::DispVec dv(sz);
  NEWMAT::Matrix StS = dv.GetS_Matrix(false);
  StS = lambda*(StS.t()*StS);
  thrust::host_vector<float> hStS(sqr(sz),0.0);
  copy_matrix(StS,thrust::raw_pointer_cast(hStS.data()));
  dStS = hStS;
} EddyCatch

} // End namespace EDDY
